#include "hip/hip_runtime.h"
#include "config.h"
#include "mesh_constants_cuda.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>


__global__ void compute_forces(int nb_blocks_to_compute,
                           int NGLOB,
                           int* d_ibool,
                           int* d_phase_ispec_inner_elastic, int num_phase_ispec_elastic,
                           int d_iphase,
                           realw* d_displ,realw* d_veloc,realw* d_accel,
                           realw* d_xix, realw* d_xiy, realw* d_xiz,
                           realw* d_etax, realw* d_etay, realw* d_etaz,
                           realw* d_gammax, realw* d_gammay, realw* d_gammaz,
                           realw* d_hprime_xx,
                           realw* d_hprimewgll_xx,
                           realw* d_wgllwgll_xy,realw* d_wgllwgll_xz,realw* d_wgllwgll_yz,
                           realw* d_kappav, realw* d_muv,
                           int NSPEC,
                           realw* d_rhostore,
                           realw* wgll_cube, 
						   bool* maskx,
						   bool* maskax,
						   int myrank){


// elastic compute kernel without attenuation
// holds for: ATTENUATION = .false.
//            COMPUTE_AND_STORE_STRAIN = .true. or .false. (true for kernel simulations)

  int bx = blockIdx.y*gridDim.x+blockIdx.x;
  int tx = threadIdx.x;

  const int NGLL3_ALIGN = NGLL3_PADDED;

  int K = (tx/NGLL2);
  int J = ((tx-K*NGLL2)/NGLLX);
  int I = (tx-K*NGLL2-J*NGLLX);

  int active,offset;
  int iglob = 0;
  int working_element;

  realw tempx1l,tempx2l,tempx3l,tempy1l,tempy2l,tempy3l,tempz1l,tempz2l,tempz3l;
  realw xixl,xiyl,xizl,etaxl,etayl,etazl,gammaxl,gammayl,gammazl,jacobianl;
  realw duxdxl,duxdyl,duxdzl,duydxl,duydyl,duydzl,duzdxl,duzdyl,duzdzl;
  realw duxdxl_plus_duydyl,duxdxl_plus_duzdzl,duydyl_plus_duzdzl;
  realw duxdyl_plus_duydxl,duzdxl_plus_duxdzl,duzdyl_plus_duydzl;

  realw fac1,fac2,fac3,lambdal,mul,lambdalplus2mul,kappal;
  realw sigma_xx,sigma_yy,sigma_zz,sigma_xy,sigma_xz,sigma_yz;

  realw sum_terms1,sum_terms2,sum_terms3;

  // gravity variables
  realw sigma_yx,sigma_zx,sigma_zy;


  __shared__ realw s_dummyx_loc[NGLL3];
  __shared__ realw s_dummyy_loc[NGLL3];
  __shared__ realw s_dummyz_loc[NGLL3];

  __shared__ realw s_tempx1[NGLL3];
  __shared__ realw s_tempx2[NGLL3];
  __shared__ realw s_tempx3[NGLL3];

  __shared__ realw s_tempy1[NGLL3];
  __shared__ realw s_tempy2[NGLL3];
  __shared__ realw s_tempy3[NGLL3];

  __shared__ realw s_tempz1[NGLL3];
  __shared__ realw s_tempz2[NGLL3];
  __shared__ realw s_tempz3[NGLL3];

  //__shared__ realw sh_hprime_xx[NGLL2];


// use only NGLL^3 = 125 active threads, plus 3 inactive/ghost threads,
// because we used memory padding from NGLL^3 = 125 to 128 to get coalescent memory accesses
  active = (tx < NGLL3 && bx < nb_blocks_to_compute) ? 1:0;

//  printf("\nwe are here: tx = %d\n",tx);
// copy from global memory to shared memory
// each thread writes one of the NGLL^3 = 125 data points
  if (active) {

    working_element = d_phase_ispec_inner_elastic[bx + num_phase_ispec_elastic*(d_iphase-1)]-1;
    iglob = d_ibool[working_element*NGLL3 + tx]-1;
    // debug
    //if( iglob < 0 || iglob >= NGLOB ){ printf("wrong iglob %d\n",iglob);  }

    // changing iglob indexing to match fortran row changes fast style
    s_dummyx_loc[tx] = d_displ[iglob*3];
    s_dummyy_loc[tx] = d_displ[iglob*3 + 1];
    s_dummyz_loc[tx] = d_displ[iglob*3 + 2];
	s_dummyx_loc[tx] = s_dummyx_loc[tx] * maskx[iglob*3];
	s_dummyy_loc[tx] = s_dummyy_loc[tx] * maskx[iglob*3 + 1];
	s_dummyz_loc[tx] = s_dummyz_loc[tx] * maskx[iglob*3 + 2];

  // JC JC here we will need to add GPU support for the new C-PML routines

  }


  __syncthreads();

  if (active) {




    tempx1l = s_dummyx_loc[K*NGLL2+J*NGLLX]*d_hprime_xx[I]
            + s_dummyx_loc[K*NGLL2+J*NGLLX+1]*d_hprime_xx[NGLLX+I]
            + s_dummyx_loc[K*NGLL2+J*NGLLX+2]*d_hprime_xx[2*NGLLX+I]
            + s_dummyx_loc[K*NGLL2+J*NGLLX+3]*d_hprime_xx[3*NGLLX+I]
            + s_dummyx_loc[K*NGLL2+J*NGLLX+4]*d_hprime_xx[4*NGLLX+I];

    tempy1l = s_dummyy_loc[K*NGLL2+J*NGLLX]*d_hprime_xx[I]
            + s_dummyy_loc[K*NGLL2+J*NGLLX+1]*d_hprime_xx[NGLLX+I]
            + s_dummyy_loc[K*NGLL2+J*NGLLX+2]*d_hprime_xx[2*NGLLX+I]
            + s_dummyy_loc[K*NGLL2+J*NGLLX+3]*d_hprime_xx[3*NGLLX+I]
            + s_dummyy_loc[K*NGLL2+J*NGLLX+4]*d_hprime_xx[4*NGLLX+I];

    tempz1l = s_dummyz_loc[K*NGLL2+J*NGLLX]*d_hprime_xx[I]
            + s_dummyz_loc[K*NGLL2+J*NGLLX+1]*d_hprime_xx[NGLLX+I]
            + s_dummyz_loc[K*NGLL2+J*NGLLX+2]*d_hprime_xx[2*NGLLX+I]
            + s_dummyz_loc[K*NGLL2+J*NGLLX+3]*d_hprime_xx[3*NGLLX+I]
            + s_dummyz_loc[K*NGLL2+J*NGLLX+4]*d_hprime_xx[4*NGLLX+I];

    tempx2l = s_dummyx_loc[K*NGLL2+I]*d_hprime_xx[J]
            + s_dummyx_loc[K*NGLL2+NGLLX+I]*d_hprime_xx[NGLLX+J]
            + s_dummyx_loc[K*NGLL2+2*NGLLX+I]*d_hprime_xx[2*NGLLX+J]
            + s_dummyx_loc[K*NGLL2+3*NGLLX+I]*d_hprime_xx[3*NGLLX+J]
            + s_dummyx_loc[K*NGLL2+4*NGLLX+I]*d_hprime_xx[4*NGLLX+J];

    tempy2l = s_dummyy_loc[K*NGLL2+I]*d_hprime_xx[J]
            + s_dummyy_loc[K*NGLL2+NGLLX+I]*d_hprime_xx[NGLLX+J]
            + s_dummyy_loc[K*NGLL2+2*NGLLX+I]*d_hprime_xx[2*NGLLX+J]
            + s_dummyy_loc[K*NGLL2+3*NGLLX+I]*d_hprime_xx[3*NGLLX+J]
            + s_dummyy_loc[K*NGLL2+4*NGLLX+I]*d_hprime_xx[4*NGLLX+J];

    tempz2l = s_dummyz_loc[K*NGLL2+I]*d_hprime_xx[J]
            + s_dummyz_loc[K*NGLL2+NGLLX+I]*d_hprime_xx[NGLLX+J]
            + s_dummyz_loc[K*NGLL2+2*NGLLX+I]*d_hprime_xx[2*NGLLX+J]
            + s_dummyz_loc[K*NGLL2+3*NGLLX+I]*d_hprime_xx[3*NGLLX+J]
            + s_dummyz_loc[K*NGLL2+4*NGLLX+I]*d_hprime_xx[4*NGLLX+J];

    tempx3l = s_dummyx_loc[J*NGLLX+I]*d_hprime_xx[K]
            + s_dummyx_loc[NGLL2+J*NGLLX+I]*d_hprime_xx[NGLLX+K]
            + s_dummyx_loc[2*NGLL2+J*NGLLX+I]*d_hprime_xx[2*NGLLX+K]
            + s_dummyx_loc[3*NGLL2+J*NGLLX+I]*d_hprime_xx[3*NGLLX+K]
            + s_dummyx_loc[4*NGLL2+J*NGLLX+I]*d_hprime_xx[4*NGLLX+K];

    tempy3l = s_dummyy_loc[J*NGLLX+I]*d_hprime_xx[K]
            + s_dummyy_loc[NGLL2+J*NGLLX+I]*d_hprime_xx[NGLLX+K]
            + s_dummyy_loc[2*NGLL2+J*NGLLX+I]*d_hprime_xx[2*NGLLX+K]
            + s_dummyy_loc[3*NGLL2+J*NGLLX+I]*d_hprime_xx[3*NGLLX+K]
            + s_dummyy_loc[4*NGLL2+J*NGLLX+I]*d_hprime_xx[4*NGLLX+K];

    tempz3l = s_dummyz_loc[J*NGLLX+I]*d_hprime_xx[K]
            + s_dummyz_loc[NGLL2+J*NGLLX+I]*d_hprime_xx[NGLLX+K]
            + s_dummyz_loc[2*NGLL2+J*NGLLX+I]*d_hprime_xx[2*NGLLX+K]
            + s_dummyz_loc[3*NGLL2+J*NGLLX+I]*d_hprime_xx[3*NGLLX+K]
            + s_dummyz_loc[4*NGLL2+J*NGLLX+I]*d_hprime_xx[4*NGLLX+K];
/*  if(myrank == 31 && bx == 0) printf("\ntx = %d, s_dummyx_loc=%f\n",tx,s_dummyx_loc[tx]);
  if(myrank == 31 && bx == 0) printf("\ntx = %d, d_hprime_xx=%f\n",tx,d_hprime_xx[tx]);
  if(myrank == 31 && bx == 0) printf("\ntx = %d, tempx1l=%e,tempx2l=%e,tempx3l=%e,tempy1l=%e\n",tx,tempx1l,tempx2l,tempx3l,tempy1l);
  if(myrank == 31 && bx == 0) printf("\nNGLLX = %d, NGLL2=%d\n",NGLLX,NGLL2);
  if(myrank == 31 && bx == 0) printf("\ntx = %d, x0=%e,x0c=%e,x1=%e,x1c=%e,x2=%e,x2c=%e,x3=%e,x3c=%e,x4=%e,x4c=%e\n",tx,
s_dummyx_loc[K*NGLL2+J*NGLLX],d_hprime_xx[I],
s_dummyx_loc[K*NGLL2+J*NGLLX+1],d_hprime_xx[NGLLX+I],
s_dummyx_loc[K*NGLL2+J*NGLLX+2],d_hprime_xx[2*NGLLX+I],
s_dummyx_loc[K*NGLL2+J*NGLLX+3],d_hprime_xx[3*NGLLX+I],
s_dummyx_loc[K*NGLL2+J*NGLLX+4],d_hprime_xx[4*NGLLX+I]);
*/



    // JC JC here we will need to add GPU support for the new C-PML routines


// compute derivatives of ux, uy and uz with respect to x, y and z
    offset = working_element*NGLL3_ALIGN + tx;

    xixl = d_xix[offset];
    xiyl = d_xiy[offset];
    xizl = d_xiz[offset];
    etaxl = d_etax[offset];
    etayl = d_etay[offset];
    etazl = d_etaz[offset];
    gammaxl = d_gammax[offset];
    gammayl = d_gammay[offset];
    gammazl = d_gammaz[offset];

    duxdxl = xixl*tempx1l + etaxl*tempx2l + gammaxl*tempx3l;
    duxdyl = xiyl*tempx1l + etayl*tempx2l + gammayl*tempx3l;
    duxdzl = xizl*tempx1l + etazl*tempx2l + gammazl*tempx3l;

    duydxl = xixl*tempy1l + etaxl*tempy2l + gammaxl*tempy3l;
    duydyl = xiyl*tempy1l + etayl*tempy2l + gammayl*tempy3l;
    duydzl = xizl*tempy1l + etazl*tempy2l + gammazl*tempy3l;

    duzdxl = xixl*tempz1l + etaxl*tempz2l + gammaxl*tempz3l;
    duzdyl = xiyl*tempz1l + etayl*tempz2l + gammayl*tempz3l;
    duzdzl = xizl*tempz1l + etazl*tempz2l + gammazl*tempz3l;

	//for dbg
/*	if(myrank == 31 && iglob == 1) printf("gpu info: %d: duxdxl= %f",iglob,duxdxl);*/ 
    // JC JC here we will need to add GPU support for the new C-PML routines

    // precompute some sums to save CPU time
    duxdxl_plus_duydyl = duxdxl + duydyl;
    duxdxl_plus_duzdzl = duxdxl + duzdzl;
    duydyl_plus_duzdzl = duydyl + duzdzl;
    duxdyl_plus_duydxl = duxdyl + duydxl;
    duzdxl_plus_duxdzl = duzdxl + duxdzl;
    duzdyl_plus_duydzl = duzdyl + duydzl;

    // JC JC here we will need to add GPU support for the new C-PML routines

    // computes deviatoric strain for kernel calculations

    // compute elements with an elastic isotropic rheology
    kappal = d_kappav[offset];
    mul = d_muv[offset];

    // full anisotropic case, stress calculations
      // isotropic case

      lambdalplus2mul = kappal + 1.33333333333333333333f * mul;  // 4./3. = 1.3333333
      lambdal = lambdalplus2mul - 2.0f * mul;
/*	
	if(tx == 4 && myrank == 31 && bx == 0) printf("\n4__stempx1: %e,%e,%e,%e,%e\n",lambdal,mul,duxdxl,duydyl,duzdzl);
	if(tx == 3 && myrank == 31 && bx == 0) printf("\n3__stempx1: %e,%e,%e,%e,%e\n",lambdal,mul,duxdxl,duydyl,duzdzl);
*/

      // compute the six components of the stress tensor sigma
      sigma_xx = lambdalplus2mul*duxdxl + lambdal*duydyl_plus_duzdzl;
      sigma_yy = lambdalplus2mul*duydyl + lambdal*duxdxl_plus_duzdzl;
      sigma_zz = lambdalplus2mul*duzdzl + lambdal*duxdxl_plus_duydyl;

      sigma_xy = mul*duxdyl_plus_duydxl;
      sigma_xz = mul*duzdxl_plus_duxdzl;
      sigma_yz = mul*duzdyl_plus_duydzl;

    jacobianl = 1.0f / (xixl*(etayl*gammazl-etazl*gammayl)-xiyl*(etaxl*gammazl-etazl*gammaxl)+xizl*(etaxl*gammayl-etayl*gammaxl));

/*	if(myrank == 31 && iglob == 1) printf("gpu:jacobian:%f,I,%d,J,%d,K,%d,element,%d,iphase,%d",jacobianl,I,J,K,working_element,d_iphase);*/
    // define symmetric components (needed for non-symmetric dot product and sigma for gravity)
    sigma_yx = sigma_xy;
    sigma_zx = sigma_xz;
    sigma_zy = sigma_yz;

    // form dot product with test vector, non-symmetric form
    s_tempx1[tx] = jacobianl * (sigma_xx*xixl + sigma_yx*xiyl + sigma_zx*xizl);
    s_tempy1[tx] = jacobianl * (sigma_xy*xixl + sigma_yy*xiyl + sigma_zy*xizl);
    s_tempz1[tx] = jacobianl * (sigma_xz*xixl + sigma_yz*xiyl + sigma_zz*xizl);

    s_tempx2[tx] = jacobianl * (sigma_xx*etaxl + sigma_yx*etayl + sigma_zx*etazl);
    s_tempy2[tx] = jacobianl * (sigma_xy*etaxl + sigma_yy*etayl + sigma_zy*etazl);
    s_tempz2[tx] = jacobianl * (sigma_xz*etaxl + sigma_yz*etayl + sigma_zz*etazl);

    s_tempx3[tx] = jacobianl * (sigma_xx*gammaxl + sigma_yx*gammayl + sigma_zx*gammazl);
    s_tempy3[tx] = jacobianl * (sigma_xy*gammaxl + sigma_yy*gammayl + sigma_zy*gammazl);
    s_tempz3[tx] = jacobianl * (sigma_xz*gammaxl + sigma_yz*gammayl + sigma_zz*gammazl);

  }

// synchronize all the threads (one thread for each of the NGLL grid points of the
// current spectral element) because we need the whole element to be ready in order
// to be able to compute the matrix products along cut planes of the 3D element below
  __syncthreads();

  // JC JC here we will need to add GPU support for the new C-PML routines

  if (active) {

    tempx1l = s_tempx1[K*NGLL2+J*NGLLX]*d_hprimewgll_xx[I*NGLLX]
            + s_tempx1[K*NGLL2+J*NGLLX+1]*d_hprimewgll_xx[I*NGLLX+1]
            + s_tempx1[K*NGLL2+J*NGLLX+2]*d_hprimewgll_xx[I*NGLLX+2]
            + s_tempx1[K*NGLL2+J*NGLLX+3]*d_hprimewgll_xx[I*NGLLX+3]
            + s_tempx1[K*NGLL2+J*NGLLX+4]*d_hprimewgll_xx[I*NGLLX+4];

    tempy1l = s_tempy1[K*NGLL2+J*NGLLX]*d_hprimewgll_xx[I*NGLLX]
            + s_tempy1[K*NGLL2+J*NGLLX+1]*d_hprimewgll_xx[I*NGLLX+1]
            + s_tempy1[K*NGLL2+J*NGLLX+2]*d_hprimewgll_xx[I*NGLLX+2]
            + s_tempy1[K*NGLL2+J*NGLLX+3]*d_hprimewgll_xx[I*NGLLX+3]
            + s_tempy1[K*NGLL2+J*NGLLX+4]*d_hprimewgll_xx[I*NGLLX+4];

    tempz1l = s_tempz1[K*NGLL2+J*NGLLX]*d_hprimewgll_xx[I*NGLLX]
            + s_tempz1[K*NGLL2+J*NGLLX+1]*d_hprimewgll_xx[I*NGLLX+1]
            + s_tempz1[K*NGLL2+J*NGLLX+2]*d_hprimewgll_xx[I*NGLLX+2]
            + s_tempz1[K*NGLL2+J*NGLLX+3]*d_hprimewgll_xx[I*NGLLX+3]
            + s_tempz1[K*NGLL2+J*NGLLX+4]*d_hprimewgll_xx[I*NGLLX+4];

    tempx2l = s_tempx2[K*NGLL2+I]*d_hprimewgll_xx[J*NGLLX]
            + s_tempx2[K*NGLL2+NGLLX+I]*d_hprimewgll_xx[J*NGLLX+1]
            + s_tempx2[K*NGLL2+2*NGLLX+I]*d_hprimewgll_xx[J*NGLLX+2]
            + s_tempx2[K*NGLL2+3*NGLLX+I]*d_hprimewgll_xx[J*NGLLX+3]
            + s_tempx2[K*NGLL2+4*NGLLX+I]*d_hprimewgll_xx[J*NGLLX+4];

    tempy2l = s_tempy2[K*NGLL2+I]*d_hprimewgll_xx[J*NGLLX]
            + s_tempy2[K*NGLL2+NGLLX+I]*d_hprimewgll_xx[J*NGLLX+1]
            + s_tempy2[K*NGLL2+2*NGLLX+I]*d_hprimewgll_xx[J*NGLLX+2]
            + s_tempy2[K*NGLL2+3*NGLLX+I]*d_hprimewgll_xx[J*NGLLX+3]
            + s_tempy2[K*NGLL2+4*NGLLX+I]*d_hprimewgll_xx[J*NGLLX+4];

    tempz2l = s_tempz2[K*NGLL2+I]*d_hprimewgll_xx[J*NGLLX]
            + s_tempz2[K*NGLL2+NGLLX+I]*d_hprimewgll_xx[J*NGLLX+1]
            + s_tempz2[K*NGLL2+2*NGLLX+I]*d_hprimewgll_xx[J*NGLLX+2]
            + s_tempz2[K*NGLL2+3*NGLLX+I]*d_hprimewgll_xx[J*NGLLX+3]
            + s_tempz2[K*NGLL2+4*NGLLX+I]*d_hprimewgll_xx[J*NGLLX+4];

    tempx3l = s_tempx3[J*NGLLX+I]*d_hprimewgll_xx[K*NGLLX]
            + s_tempx3[NGLL2+J*NGLLX+I]*d_hprimewgll_xx[K*NGLLX+1]
            + s_tempx3[2*NGLL2+J*NGLLX+I]*d_hprimewgll_xx[K*NGLLX+2]
            + s_tempx3[3*NGLL2+J*NGLLX+I]*d_hprimewgll_xx[K*NGLLX+3]
            + s_tempx3[4*NGLL2+J*NGLLX+I]*d_hprimewgll_xx[K*NGLLX+4];

    tempy3l = s_tempy3[J*NGLLX+I]*d_hprimewgll_xx[K*NGLLX]
            + s_tempy3[NGLL2+J*NGLLX+I]*d_hprimewgll_xx[K*NGLLX+1]
            + s_tempy3[2*NGLL2+J*NGLLX+I]*d_hprimewgll_xx[K*NGLLX+2]
            + s_tempy3[3*NGLL2+J*NGLLX+I]*d_hprimewgll_xx[K*NGLLX+3]
            + s_tempy3[4*NGLL2+J*NGLLX+I]*d_hprimewgll_xx[K*NGLLX+4];

    tempz3l = s_tempz3[J*NGLLX+I]*d_hprimewgll_xx[K*NGLLX]
            + s_tempz3[NGLL2+J*NGLLX+I]*d_hprimewgll_xx[K*NGLLX+1]
            + s_tempz3[2*NGLL2+J*NGLLX+I]*d_hprimewgll_xx[K*NGLLX+2]
            + s_tempz3[3*NGLL2+J*NGLLX+I]*d_hprimewgll_xx[K*NGLLX+3]
            + s_tempz3[4*NGLL2+J*NGLLX+I]*d_hprimewgll_xx[K*NGLLX+4];


    fac1 = d_wgllwgll_yz[K*NGLLX+J];
    fac2 = d_wgllwgll_xz[K*NGLLX+I];
    fac3 = d_wgllwgll_xy[J*NGLLX+I];

    sum_terms1 = - (fac1*tempx1l + fac2*tempx2l + fac3*tempx3l) * maskax[iglob*3];
    sum_terms2 = - (fac1*tempy1l + fac2*tempy2l + fac3*tempy3l) * maskax[iglob*3 + 1];
    sum_terms3 = - (fac1*tempz1l + fac2*tempz2l + fac3*tempz3l) * maskax[iglob*3 + 2];

    // adds gravity term
      atomicAdd(&d_accel[iglob*3], sum_terms1);
      atomicAdd(&d_accel[iglob*3+1], sum_terms2);
      atomicAdd(&d_accel[iglob*3+2], sum_terms3);
/*	  if(iglob == 1 && myrank == 31) printf("\ngpu : delta: %f\n",sum_terms1);
	  if(iglob == 1 && myrank == 31) printf("\ngpu : fac1 %f tempx1l%f\n, s_tempx1,%f,tx= %d\n",fac1,tempx1l,s_tempx1[tx],tx);

	  if(iglob == 1 && myrank == 31)
	  {
	  for(int ii = 0;ii<125;ii++) printf("\nstemp%d: = %f\n",ii,s_tempx1[ii]);
	  }
	  */
  } // if(active)
} // kernel_2_noatt_impl()


extern "C"
void FC_FUNC_(compute_forces_fault,
		      COMPUTE_FORCES_FAULT)(long* Mesh_pointer,
                                    int* iphase,
                                    realw* deltat,
									realw* CG_d_displ,
									realw* CG_d_accel,
									bool* maskx,
									bool* maskax,
                                    int* nspec_outer_elastic,
                                    int* nspec_inner_elastic,
									int* myrank )
{

  TRACE("\tcompute_forces_fault");
  // EPIK_TRACER("compute_forces_viscoelastic_cuda");
  //printf("Running compute_forces\n");
  //double start_time = get_time();

  Mesh* mp = (Mesh*)(*Mesh_pointer); // get Mesh from fortran integer wrapper

  int num_elements;

  if( *iphase == 1 )
    num_elements = *nspec_outer_elastic;
  else
    num_elements = *nspec_inner_elastic;

  // checks if anything to do
  if( num_elements == 0 ) return;

  int blocksize = NGLL3_PADDED;

  int num_blocks_x, num_blocks_y;
  get_blocks_xy(num_elements,&num_blocks_x,&num_blocks_y);

  dim3 grid(num_blocks_x,num_blocks_y);
  dim3 threads(blocksize,1,1);

  compute_forces<<<grid,threads>>>(
		  num_elements,
		  mp->NGLOB_AB,
		  mp->d_ibool,
		  mp->d_phase_ispec_inner_elastic,
		  mp->num_phase_ispec_elastic,
		  *iphase,
		  CG_d_displ,	mp->d_veloc,	CG_d_accel,
		  mp->d_xix,	mp->d_xiy,		mp->d_xiz,
		  mp->d_etax,	mp->d_etay,		mp->d_etaz,
		  mp->d_gammax,	mp->d_gammay,	mp->d_gammaz,
		  mp->d_hprime_xx,mp->d_hprimewgll_xx,
		  mp->d_wgllwgll_xy,
		  mp->d_wgllwgll_xz,
		  mp->d_wgllwgll_yz,
		  mp->d_kappav,	mp->d_muv,
		  mp->NSPEC_AB,
		  mp->d_rhostore,
		  mp->d_wgll_cube,
		  maskx,
		  maskax,
		  *myrank);

}
